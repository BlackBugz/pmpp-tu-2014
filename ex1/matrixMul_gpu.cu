#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

#include "common.h"
#include "matrix.h"
#include "matrixMul_gpu.h"

// -------------------------------------------------------------------------------
// Simple implementation, no memory enhancements

__global__ void MatrixMulKernelSimple(Matrix M, Matrix N, Matrix P);

void MatrixMulGPUSimple(const Matrix &M, const Matrix &N, Matrix &P)
{
	// TODO Task 4: Determine execution configuration and call CUDA kernel
	dim3 dimBlock(P.width, P.width);
	dim3 dimGrid(1,1);

	MatrixMulKernelSimple<<<dimGrid, dimBlock>>>(M, N, P);
}

// TODO Task 4: Implement matrix multiplication CUDA kernel

/** \brief Kernel function to perform multiplication on the device
 *  \param M First input matrix
 *  \param N Second input matrix
 *  \param P Output matrix
 */
__global__ void MatrixMulKernelSimple(Matrix M, Matrix N, Matrix P)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float p = 0;

	for(int i = 0; i < M.height; ++i){
		float m = M.elements[ty * M.pitch + i];
		float n = N.elements[i * N.pitch + tx];
		p += m*n;
	}

	P.elements[ty * P.pitch + tx] = p;
}


// -----------------------------------------------------------------------------------
// Tiled multiplication

__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P);

void MatrixMulGPU(const Matrix &M, const Matrix &N, Matrix &P)
{
	// TODO Task 4: Determine execution configuration and call CUDA kernel
	dim3 dimBlock(P.width, P.width);
	dim3 dimGrid(1,1);

	MatrixMulKernel<<<dimGrid, dimBlock>>>(M, N, P);
}

// TODO Task 4: Implement matrix multiplication CUDA kernel

/** \brief Kernel function to perform multiplication on the device
 *  \param M First input matrix
 *  \param N Second input matrix
 *  \param P Output matrix
 */
__global__ void MatrixMulKernel(Matrix M, Matrix N, Matrix P)
{
	int tx = threadIdx.x;
	int ty = threadIdx.y;

	float p = 0;

	for(int i = 0; i < M.height; ++i){
		float m = M.elements[ty * M.pitch + i];
		float n = N.elements[i * N.pitch + tx];
		p += m*n;
	}

	P.elements[ty * P.pitch + tx] = p;
}
