#include "hip/hip_runtime.h"
/* *
 * Copyright 1993-2012 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 */
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>

#include "common.h"
#include "ppm.h"

#include "convolution_gpu_tmem.h"

#define BLOCK_SIZE 32

texture<unsigned int, hipTextureType1D, hipReadModeElementType> texRefImg;

__global__ void ConvolveHGPUTMem(unsigned int *dst, const unsigned int *src, const float *kernel, int kernelSize, int w, int h)
{

	int bx = blockIdx.x;
	int by = blockIdx.y;

	int tx = threadIdx.x;
	int ty = threadIdx.y;

	int row = by * blockDim.y + ty;
	int col = bx * blockDim.x + tx;

	float finalRed = 0.0f;
	float finalGreen = 0.0f;
	float finalBlue = 0.0f;

	for (int i = 0; i < kernelSize; i++)
	{
		int px = col + (i - kernelSize/2);

		// Clamp to [0, w-1]
		px = MIN(px, w-1);
		px = MAX(px, 0);

		//unsigned int pixel = src[row * w + px];
		unsigned int pixel = tex1Dfetch(texRefImg, row * w + px);

		unsigned char r = pixel & 0x000000ff;
		unsigned char g = (pixel & 0x0000ff00) >> 8;
		unsigned char b = (pixel & 0x00ff0000) >> 16;

		finalRed   += r * kernel[i];
		finalGreen += g * kernel[i];
		finalBlue  += b * kernel[i];
	}

	unsigned char finalRed_uc = roundf(finalRed);
	unsigned char finalGreen_uc = roundf(finalGreen);
	unsigned char finalBlue_uc = roundf(finalBlue);

	unsigned int finalPixel = finalRed_uc
		| (finalGreen_uc << 8)
		| (finalBlue_uc << 16);
	dst[row * w + col] = finalPixel;


}

__global__ void ConvolveVGPUTMem(unsigned int *dst, const float *kernel, int kernelSize, int w, int h)
{
		int bx = blockIdx.x;
		int by = blockIdx.y;

		int tx = threadIdx.x;
		int ty = threadIdx.y;

		int row = by * blockDim.y + ty;
		int col = bx * blockDim.x + tx;

		float finalRed = 0.0f;
		float finalGreen = 0.0f;
		float finalBlue = 0.0f;

		for (int i = 0; i < kernelSize; i++)
		{
			int py = row + (i - kernelSize/2);

			// Clamp to [0, h-1]
			py = MIN(py, h-1);
			py = MAX(py, 0);

			unsigned int pixel = tex1Dfetch(texRefImg, py * w + col);

			unsigned char r = pixel & 0x000000ff;
			unsigned char g = (pixel & 0x0000ff00) >> 8;
			unsigned char b = (pixel & 0x00ff0000) >> 16;

			finalRed   += r * kernel[i];
			finalGreen += g * kernel[i];
			finalBlue  += b * kernel[i];
		}

		unsigned char finalRed_uc = roundf(finalRed);
		unsigned char finalGreen_uc = roundf(finalGreen);
		unsigned char finalBlue_uc = roundf(finalBlue);

		unsigned int finalPixel = finalRed_uc
			| (finalGreen_uc << 8)
			| (finalBlue_uc << 16);
		dst[row * w + col] = finalPixel;
}

void ApplyFilterGPUTMem(PPMImage &destImg, PPMImage &srcImg, const float * kernel, unsigned int kernelSize)
{
	CUDA_SUCCEEDED(hipBindTexture(0, texRefImg, srcImg.data, srcImg.height*srcImg.width*sizeof(unsigned int)));

	dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
	dim3 dimGrid(divUp(srcImg.width,BLOCK_SIZE),divUp(srcImg.height,BLOCK_SIZE));

	ConvolveHGPUTMem<<<dimGrid, dimBlock>>>(destImg.data, srcImg.data, kernel, kernelSize, srcImg.width, srcImg.height);

	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}

	CUDA_SUCCEEDED(hipUnbindTexture(texRefImg));

	unsigned int * bk;
	bk = srcImg.data;
	srcImg.data = destImg.data;
	destImg.data =bk;

	CUDA_SUCCEEDED(hipBindTexture(0, texRefImg, srcImg.data, destImg.height*destImg.width*sizeof(unsigned int)));

	ConvolveVGPUTMem<<<dimGrid, dimBlock>>>(destImg.data, kernel, kernelSize, srcImg.width, srcImg.height);
	error = hipGetLastError();
	if(error != hipSuccess)
	{
		// print the CUDA error message and exit
		printf("CUDA error: %s\n", hipGetErrorString(error));
	}
	CUDA_SUCCEEDED(hipUnbindTexture(texRefImg));

}
